/* -*- C++ -*- */
/*    This file is part of CUDA_eternity2
      Copyright (C) 2016  Julien Thevenon ( julien_thevenon at yahoo.fr )

      This program is free software: you can redistribute it and/or modify
      it under the terms of the GNU General Public License as published by
      the Free Software Foundation, either version 3 of the License, or
      (at your option) any later version.

      This program is distributed in the hope that it will be useful,
      but WITHOUT ANY WARRANTY; without even the implied warranty of
      MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
      GNU General Public License for more details.

      You should have received a copy of the GNU General Public License
      along with this program.  If not, see <http://www.gnu.org/licenses/>
*/
#include "my_cuda.h"
#include "border_backtracker.h"
#include "border_pieces.h"
#include "border_color_constraint.h"
#include "border_constraint_generator.h"
#include "octet_array.h"
#include <iostream>
#include <sstream>
#include <iomanip>
#include <string>

CUDA_KERNEL(border_backtracker_kernel,
	    const border_pieces & p_border_pieces,
	    border_color_constraint  (&p_border_constraints)[23],
	    octet_array * p_initial_constraint
	    )
{
  unsigned int l_index = 0;
  border_color_constraint l_available_pieces(true);
  octet_array l_solution;
  bool l_ended = false;
  do
    {
      unsigned int l_previous_index = l_index ? l_index - 1 : 59;
      unsigned int l_piece_id = l_solution.get_octet(l_previous_index);
      unsigned int l_color =  l_piece_id ? p_border_pieces.get_right(l_piece_id - 1) : 0;
      border_color_constraint l_available_transitions = p_border_constraints[l_color];
      l_available_transitions & p_border_constraints[p_initial_constraint[threadIdx.x + blockIdx.x * blockDim.x].get_octet(l_index)];
      unsigned int l_next_index = l_index < 59 ? l_index + 1 : 0;
      uint64_t l_corner_mask = (0 == l_index || 15 == l_index || 30 == l_index || 45 == l_index) ? 0xF : UINT64_MAX;
      l_available_transitions & l_corner_mask;
      l_available_transitions & l_available_pieces;
      l_available_transitions & (~(( ((uint64_t)1) << l_solution.get_octet(l_index)) - 1));

      int l_ffs = l_available_transitions.ffs();

      // Detect the end in case we have found no solution ( index 0 and no candidate)
      // or in case we are at the end ( next_index = 0 and there is one candidate)
      l_ended = (!l_index && !l_ffs) || (!l_next_index && l_ffs);

      // Remove the piece from list of available pieces if a transition was
      // possible or restablish it to prepare come back to previous state
      unsigned int l_toggled_index = l_ffs ? l_ffs : l_solution.get_octet(l_previous_index);
      l_available_pieces.toggle_bit(l_toggled_index - 1,true);

      // Prepare for next pieces
      l_solution.set_octet(l_index, l_ffs);
      l_index = l_ffs ? l_next_index : l_previous_index;
 
    }
  while(!l_ended);
  p_initial_constraint[threadIdx.x + blockIdx.x * blockDim.x] = l_solution;
}

//------------------------------------------------------------------------------
int launch_border_bactracker(unsigned int p_nb_cases,
			     unsigned int p_nb_block,
			     unsigned int p_nb_thread,
			     const std::string & p_initial_situation,
			     const border_pieces & p_border_pieces,
			     border_color_constraint  (&p_border_constraints)[23],
			     const unsigned int (&p_border_edges)[60],
			     const std::map<unsigned int, unsigned int> & p_B2C_color_count,
			     const std::map<unsigned int, unsigned int> & p_reorganised_colors
			     )
{
  gpuErrChk(hipDeviceSetCacheConfig(hipFuncCachePreferL1)); // type hipFuncCache_t
  unsigned int l_block_size = p_nb_thread;
  std::cout << "Nb cases : " << p_nb_cases << std::endl;
  std::cout << "Nb blocks : " << p_nb_block << std::endl;
  std::cout << "Block_size : " << l_block_size << " threads" << std::endl;
  unsigned int l_nb_constraints = l_block_size * p_nb_block;
  octet_array * l_initial_constraint = new octet_array[l_nb_constraints];

  std::string l_situation_string = p_initial_situation;

  border_constraint_generator l_generator(p_B2C_color_count);

  // Prepare pointers for memory allocation on GPU
  octet_array * l_initial_constraint_ptr = nullptr;
  border_pieces * l_border_pieces_ptr = nullptr;
  border_color_constraint  (* l_border_constraints_ptr)[23] = nullptr;

  // Allocate pointers on GPU
  gpuErrChk(hipMalloc(&l_initial_constraint_ptr, l_nb_constraints * sizeof(octet_array)));
  gpuErrChk(hipMalloc(&l_border_pieces_ptr, sizeof(border_pieces)));
  gpuErrChk(hipMalloc(&l_border_constraints_ptr, 23 * sizeof(border_color_constraint)));

  gpuErrChk(hipMemcpy(l_border_pieces_ptr, &p_border_pieces, sizeof(border_pieces), hipMemcpyHostToDevice));
  gpuErrChk(hipMemcpy(l_border_constraints_ptr, &p_border_constraints[0], 23 * sizeof(border_color_constraint), hipMemcpyHostToDevice));

  bool l_found = false;
  uint64_t l_fail_counter = 0;
  unsigned int l_nb_loop = 0;
  while(!l_found && l_fail_counter < p_nb_cases)
    {
      for(unsigned int l_index = 0; l_index < l_nb_constraints; ++l_index)
	{
	  l_generator.generate(l_initial_constraint[l_index]);

	  std::map<unsigned int,unsigned int> l_check;
	  for(unsigned int l_octet = 0; l_octet < 60; ++l_octet)
	    {
#if 0
	      std::cout << std::setw(2) << l_initial_constraint[l_index].get_octet(l_octet) << " " ;
#endif
	      if(l_initial_constraint[l_index].get_octet(l_octet))
		{
		  l_check[l_initial_constraint[l_index].get_octet(l_octet)]++;
		}
	    }
#if 0
	  std::cout << std::endl ;
#endif
	  assert(l_check == p_B2C_color_count);
	  if("" != l_situation_string)
	    {
	      extract_initial_constraint(l_situation_string,
					 l_initial_constraint[l_index],
					 p_border_pieces
					 );
	    }
	}

      gpuErrChk(hipMemcpy(l_initial_constraint_ptr, &l_initial_constraint[0], l_nb_constraints * sizeof(octet_array), hipMemcpyHostToDevice));

      dim3 dimBlock(l_block_size,1);
      dim3 dimGrid(p_nb_block,1);
      launch_kernels(border_backtracker_kernel, dimGrid, dimBlock, *l_border_pieces_ptr,
		     *l_border_constraints_ptr,
		     l_initial_constraint_ptr
		     );


      gpuErrChk(hipMemcpy(&l_initial_constraint[0], l_initial_constraint_ptr, l_nb_constraints * sizeof(octet_array), hipMemcpyDeviceToHost));

      for(unsigned int l_index = 0; l_index < l_nb_constraints ; ++l_index)
	{
	  if(l_initial_constraint[l_index].get_octet(0))
	    {
	      std::string l_result;
	      char l_orientation2string[4] = {'N', 'E', 'S', 'W'};
	      for(unsigned int l_y = 0; l_y < 16; ++l_y)
		{
		  for(unsigned int l_x = 0; l_x < 16; ++l_x)
		    {
		      std::stringstream l_stream;
		      if(0 == l_y && 0 == l_x)
			{
			  l_stream << std::setw(3) << l_initial_constraint[l_index].get_octet(0) << l_orientation2string[(p_border_edges[l_initial_constraint[l_index].get_octet(0) - 1] + 1) % 4];
			  l_result += l_stream.str();
			}
		      else if(0 == l_y && 15 == l_x)
			{
			  l_stream << std::setw(3) << l_initial_constraint[l_index].get_octet(15) << l_orientation2string[p_border_edges[l_initial_constraint[l_index].get_octet(15) - 1]];
			  l_result += l_stream.str();
			}
		      else if(15 == l_y && 15 == l_x)
			{
			  l_stream << std::setw(3) << l_initial_constraint[l_index].get_octet(30) << l_orientation2string[(p_border_edges[l_initial_constraint[l_index].get_octet(30) - 1] + 3) % 4];
			  l_result += l_stream.str();
			}
		      else if(15 == l_y && 0 == l_x)
			{
			  l_stream << std::setw(3) << l_initial_constraint[l_index].get_octet(45) << l_orientation2string[(p_border_edges[l_initial_constraint[l_index].get_octet(45) - 1] + 2) % 4];
			  l_result += l_stream.str();
			}
		      else if(0 == l_y)
			{
			  l_stream << std::setw(3) << l_initial_constraint[l_index].get_octet(l_x) << l_orientation2string[p_border_edges[l_initial_constraint[l_index].get_octet(l_x) - 1]];
			  l_result += l_stream.str();
			}
		      else if(15 == l_x)
			{
			  l_stream << std::setw(3) << l_initial_constraint[l_index].get_octet(15 + l_y) << l_orientation2string[(p_border_edges[l_initial_constraint[l_index].get_octet(l_x) - 1] + 3) % 4];
			  l_result += l_stream.str();
			}
		      else if(15 == l_y)
			{
			  l_stream << std::setw(3) << l_initial_constraint[l_index].get_octet(30 - l_x + 15) << l_orientation2string[(p_border_edges[l_initial_constraint[l_index].get_octet(l_x) - 1] + 2) % 4];
			  l_result += l_stream.str();
			}
		      else if(0 == l_x)
			{
			  l_stream << std::setw(3) << l_initial_constraint[l_index].get_octet(45 - l_y + 15) << l_orientation2string[(p_border_edges[l_initial_constraint[l_index].get_octet(l_x) - 1] + 1) % 4];
			  l_result += l_stream.str();
			}
		      else
			{
			  l_result += "----";
			}
		    }
		  //  l_result += "\n";
		}
	      std::cout << "\"" << l_result << "\"" << std::endl ;
	      l_found = true;
	    }
	  else
	    {
	      ++l_fail_counter;
	    }
	}
      ++l_nb_loop;
    }

  // Free pointers on GPU
  gpuErrChk(hipFree(l_initial_constraint_ptr));
  gpuErrChk(hipFree(l_border_pieces_ptr));
  gpuErrChk(hipFree(l_border_constraints_ptr));

  delete[] l_initial_constraint;
  std::cout << "Nb loop : " << l_nb_loop << std::endl;
  std::cout << l_fail_counter << " fails" << std::endl;
  return EXIT_SUCCESS;
}

//------------------------------------------------------------------------------
void extract_initial_constraint(const std::string & p_situation_string,
				octet_array & p_initial_constraint,
				const border_pieces & p_border_pieces
				)
{
  assert(256 * 4 == p_situation_string.size());
  for(unsigned int l_situation_index = 0 ;
      l_situation_index < 256 ;
      ++l_situation_index
      )
    {
      std::string l_piece_id_str = p_situation_string.substr(l_situation_index * 4,3);
      if("---" != l_piece_id_str)
	{
	  unsigned int l_piece_id = std::stoi(l_piece_id_str);
	  unsigned int l_constraint_index= 0;
	  bool l_meaningful = true;
	  if(l_situation_index < 16)
	    {
	      l_constraint_index = l_situation_index;
	    }
	  else if(15 == l_situation_index % 16)
	    {
	      l_constraint_index = 15 + (l_situation_index / 16);
	    }
	  else if(15 == l_situation_index / 16)
	    {
	      l_constraint_index = 255 - l_situation_index + 30;
	    }
	  else if(0 == l_situation_index % 16)
	    {
	      l_constraint_index = 45 - (l_situation_index / 16 ) + 15;
	    }
	  else
	    {
	      l_meaningful = false;
	    }
	  if(l_meaningful)
	    {
	      p_initial_constraint.set_octet(l_constraint_index, p_border_pieces.get_center(l_piece_id - 1));
	    }
	}
    }
}

// EOF
