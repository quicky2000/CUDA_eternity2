/* -*- C++ -*- */
/*    This file is part of CUDA_eternity2
      Copyright (C) 2016  Julien Thevenon ( julien_thevenon at yahoo.fr )

      This program is free software: you can redistribute it and/or modify
      it under the terms of the GNU General Public License as published by
      the Free Software Foundation, either version 3 of the License, or
      (at your option) any later version.

      This program is distributed in the hope that it will be useful,
      but WITHOUT ANY WARRANTY; without even the implied warranty of
      MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
      GNU General Public License for more details.

      You should have received a copy of the GNU General Public License
      along with this program.  If not, see <http://www.gnu.org/licenses/>
*/
#include "my_cuda.h"
#include "border_pieces.h"
#include "border_color_constraint.h"
#include "border_constraint_generator.h"
#include "octet_array.h"
#include <iostream>
#include <sstream>
#include <iomanip>
#include <string>

CUDA_KERNEL(border_backtracker_kernel, const border_pieces & p_border_pieces, border_color_constraint  (&p_border_constraints)[23], octet_array * p_initial_constraint)
{
  unsigned int l_index = 0;
  border_color_constraint l_available_pieces[60];
  l_available_pieces[59].fill(true);
  border_color_constraint l_available_transitions[60];
  l_available_transitions[0].fill(true);
  octet_array l_solution;
  bool l_ended = false;
  do
    {
      unsigned int l_previous_index = l_index ? l_index - 1 : 59;
      unsigned int l_piece_id = l_solution.get_octet(l_previous_index);
      unsigned int l_color =  l_piece_id ? p_border_pieces.get_right(l_piece_id - 1) : 0;
      l_available_transitions[l_index] & p_border_constraints[l_color];
      l_available_transitions[l_index] & p_border_constraints[p_initial_constraint[threadIdx.x].get_octet(l_index)];
      unsigned int l_next_index = l_index < 59 ? l_index + 1 : 0;
      l_piece_id = l_solution.get_octet(l_next_index);
      l_color = l_piece_id ? p_border_pieces.get_left(l_piece_id - 1) : 0;
      l_available_transitions[l_index] & p_border_constraints[l_color];
      uint64_t l_corner_mask = (0 == l_index || 15 == l_index || 30 == l_index || 45 == l_index) ? 0xF : UINT64_MAX;
      l_available_transitions[l_index] & l_corner_mask;
      l_available_transitions[l_index] & l_available_pieces[l_previous_index];
      int l_ffs = l_available_transitions[l_index].ffs();

      // Detect the end in case we have found no solution ( index 0 and no candidate)
      // or in case we are at the end ( next_index = 0 and there is one candidate)
      l_ended = (!l_index && !l_ffs) || (!l_next_index && l_ffs);

      // Apply mask to indicate the piece we will check for
      l_available_pieces[l_index] = l_available_pieces[l_previous_index];
      l_available_transitions[l_index].toggle_bit(l_ffs - 1, l_ffs);
      l_available_pieces[l_index].toggle_bit(l_ffs - 1, l_ffs);
      l_available_transitions[l_next_index].fill(true);

      // Prepare for next pieces
      l_solution.set_octet(l_index, l_ffs);
      l_index = l_ffs ? l_next_index : l_previous_index;
 
    }
  while(!l_ended);
  p_initial_constraint[threadIdx.x] = l_solution;
}

//------------------------------------------------------------------------------
int launch_border_bactracker(const border_pieces & p_border_pieces,
			     border_color_constraint  (&p_border_constraints)[23],
			     const unsigned int (&p_border_edges)[60],
			     const std::map<unsigned int, unsigned int> & p_B2C_color_count
			     )
{
  unsigned int l_block_size = 32;
  std::cout << "Block-size = " << l_block_size << std::endl;
  octet_array * l_initial_constraint = new octet_array[l_block_size];

  border_constraint_generator l_generator(p_B2C_color_count);

  bool l_found = false;
  uint64_t l_fail_counter = 0;
  unsigned int l_nb_loop = 0;
  while(!l_found && l_fail_counter < 1024 * 1024)
    {
      for(unsigned int l_index = 0; l_index < l_block_size; ++l_index)
	{
	  l_generator.generate(l_initial_constraint[l_index]);

	  std::map<unsigned int,unsigned int> l_check;
	  for(unsigned int l_octet = 0; l_octet < 60; ++l_octet)
	    {
#if 0
	      std::cout << std::setw(2) << l_initial_constraint.get_octet(l_octet) << " " ;
#endif
	      if(l_initial_constraint[l_index].get_octet(l_octet))
		{
		  l_check[l_initial_constraint[l_index].get_octet(l_octet)]++;
		}
	    }
#if 0
	  std::cout << std::endl ;
#endif
	  assert(l_check == p_B2C_color_count);
	}

      // Prepare pointers for memory allocation on GPU
      octet_array * l_initial_constraint_ptr = nullptr;
      border_pieces * l_border_pieces_ptr = nullptr;
      border_color_constraint  (* l_border_constraints_ptr)[23] = nullptr;

      // Allocate pointers on GPU
      gpuErrChk(hipMalloc(&l_initial_constraint_ptr, l_block_size * sizeof(octet_array)));
      gpuErrChk(hipMalloc(&l_border_pieces_ptr, sizeof(border_pieces)));
      gpuErrChk(hipMalloc(&l_border_constraints_ptr, 23 * sizeof(border_color_constraint)));

      gpuErrChk(hipMemcpy(l_initial_constraint_ptr, &l_initial_constraint[0], l_block_size * sizeof(octet_array), hipMemcpyHostToDevice));
      gpuErrChk(hipMemcpy(l_border_pieces_ptr, &p_border_pieces, sizeof(border_pieces), hipMemcpyHostToDevice));
      gpuErrChk(hipMemcpy(l_border_constraints_ptr, &p_border_constraints[0], 23 * sizeof(border_color_constraint), hipMemcpyHostToDevice));

      dim3 dimBlock(l_block_size,1);
      dim3 dimGrid(1,1);
      launch_kernels(border_backtracker_kernel, dimGrid, dimBlock, *l_border_pieces_ptr,
		     *l_border_constraints_ptr,
		     l_initial_constraint_ptr
		     );


      gpuErrChk(hipMemcpy(&l_initial_constraint[0], l_initial_constraint_ptr, l_block_size * sizeof(octet_array), hipMemcpyDeviceToHost));

      // Free pointers on GPU
      gpuErrChk(hipFree(l_initial_constraint_ptr));
      gpuErrChk(hipFree(l_border_pieces_ptr));
      gpuErrChk(hipFree(l_border_constraints_ptr));

      for(unsigned int l_index = 0; l_index < l_block_size; ++l_index)
	{
	  if(l_initial_constraint[l_index].get_octet(0))
	    {
	      std::string l_result;
	      char l_orientation2string[4] = {'N', 'E', 'S', 'W'};
	      for(unsigned int l_y = 0; l_y < 16; ++l_y)
		{
		  for(unsigned int l_x = 0; l_x < 16; ++l_x)
		    {
		      std::stringstream l_stream;
		      if(0 == l_y && 0 == l_x)
			{
			  l_stream << std::setw(3) << l_initial_constraint[l_index].get_octet(0) << l_orientation2string[(p_border_edges[l_initial_constraint[l_index].get_octet(0) - 1] + 1) % 4];
			  l_result += l_stream.str();
			}
		      else if(0 == l_y && 15 == l_x)
			{
			  l_stream << std::setw(3) << l_initial_constraint[l_index].get_octet(15) << l_orientation2string[p_border_edges[l_initial_constraint[l_index].get_octet(15) - 1]];
			  l_result += l_stream.str();
			}
		      else if(15 == l_y && 15 == l_x)
			{
			  l_stream << std::setw(3) << l_initial_constraint[l_index].get_octet(30) << l_orientation2string[(p_border_edges[l_initial_constraint[l_index].get_octet(30) - 1] + 3) % 4];
			  l_result += l_stream.str();
			}
		      else if(15 == l_y && 0 == l_x)
			{
			  l_stream << std::setw(3) << l_initial_constraint[l_index].get_octet(45) << l_orientation2string[(p_border_edges[l_initial_constraint[l_index].get_octet(45) - 1] + 2) % 4];
			  l_result += l_stream.str();
			}
		      else if(0 == l_y)
			{
			  l_stream << std::setw(3) << l_initial_constraint[l_index].get_octet(l_x) << l_orientation2string[p_border_edges[l_initial_constraint[l_index].get_octet(l_x) - 1]];
			  l_result += l_stream.str();
			}
		      else if(15 == l_x)
			{
			  l_stream << std::setw(3) << l_initial_constraint[l_index].get_octet(15 + l_y) << l_orientation2string[(p_border_edges[l_initial_constraint[l_index].get_octet(l_x) - 1] + 3) % 4];
			  l_result += l_stream.str();
			}
		      else if(15 == l_y)
			{
			  l_stream << std::setw(3) << l_initial_constraint[l_index].get_octet(30 - l_x + 15) << l_orientation2string[(p_border_edges[l_initial_constraint[l_index].get_octet(l_x) - 1] + 2) % 4];
			  l_result += l_stream.str();
			}
		      else if(0 == l_x)
			{
			  l_stream << std::setw(3) << l_initial_constraint[l_index].get_octet(45 - l_y + 15) << l_orientation2string[(p_border_edges[l_initial_constraint[l_index].get_octet(l_x) - 1] + 1) % 4];
			  l_result += l_stream.str();
			}
		      else
			{
			  l_result += "----";
			}
		    }
		  //  l_result += "\n";
		}
	      std::cout << "\"" << l_result << "\"" << std::endl ;
	      l_found = true;
	    }
	  else
	    {
	      ++l_fail_counter;
	    }
	}
      ++l_nb_loop;
    }
  std::cout << "Nb loop : " << l_nb_loop << std::endl;
  std::cout << l_fail_counter << " fails" << std::endl;
  return EXIT_SUCCESS;
}
// EOF
