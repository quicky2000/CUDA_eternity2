#include "hip/hip_runtime.h"
/* -*- C++ -*- */
/*    This file is part of CUDA_tests
      Copyright (C) 2016  Julien Thevenon ( julien_thevenon at yahoo.fr )

      This program is free software: you can redistribute it and/or modify
      it under the terms of the GNU General Public License as published by
      the Free Software Foundation, either version 3 of the License, or
      (at your option) any later version.

      This program is distributed in the hope that it will be useful,
      but WITHOUT ANY WARRANTY; without even the implied warranty of
      MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
      GNU General Public License for more details.

      You should have received a copy of the GNU General Public License
      along with this program.  If not, see <http://www.gnu.org/licenses/>
*/
#include <iostream>
#include <cinttypes>
#include <cstring>
#include <sstream>
#include <cassert>
#include <stdexcept>

#include "my_cuda.h"

typedef enum class orientation {NORTH=0,EAST,SOUTH,WEST} t_orientation;

class constraint
{
public:

  CUDA_METHOD_HD_I constraint(bool p_init=false);
  CUDA_METHOD_HD_I constraint(const constraint & p_constraint);

  CUDA_METHOD_HD_I void operator=(const constraint & p_constraint);

  CUDA_METHOD_HD_I constraint(const constraint & p_c1,
			      const constraint & p_c2,
			      const constraint & p_c3,
			      const constraint & p_c4,
			      const constraint & p_c5
			      );
  CUDA_METHOD_HD_I void toggle_bit(uint32_t p_index, bool p_value);

  inline void fill(bool p_init);
  inline void set_bit(uint32_t p_index);
  inline void unset_bit(uint32_t p_index);

  inline bool get_bit(uint32_t p_index) const;

  CUDA_METHOD_HD_I int ffs(void) const;
private:
  uint32_t m_words[7];
};

constraint::constraint(const constraint & p_constraint)
{
  memcpy(&m_words[0], &p_constraint.m_words[0], 7 * sizeof(uint32_t));
}

void constraint::operator=(const constraint & p_constraint)
{
 memcpy(&m_words[0], &p_constraint.m_words[0], 7 * sizeof(uint32_t));
}

inline void test_constraint(void);

class situation_orientation
{
public:
  CUDA_METHOD_HD_I situation_orientation(void);
  CUDA_METHOD_HD_I situation_orientation(const situation_orientation & p_orientation);

  CUDA_METHOD_HD_I void set_orientation(unsigned int p_index,
					uint32_t p_orientation);
  CUDA_METHOD_HD_I uint32_t get_orientation(unsigned int p_index) const;
  inline void set_orientation(unsigned int p_x_index,
			      unsigned int p_y_index,
			      uint32_t p_orientation);
  inline uint32_t get_orientation(unsigned int p_x_index,
				  unsigned int p_y_index) const;
private:
  uint32_t m_orientations[16];
};

inline void test_orientation(void);

class situation
{
public:
  situation(void) = default;
  CUDA_METHOD_HD_I situation(const situation & p_situation);
  CUDA_METHOD_HD_I void operator=(const situation & p_situation);

  situation_orientation m_orientations;
  uint8_t m_piece_ids[256];
}
;

class piece
{
public:
  inline piece(void);

  inline void set_color(uint8_t p_north_color,
			unsigned int p_orientation);

  CUDA_METHOD_HD_I uint8_t get_color(unsigned int p_side) const;

  CUDA_METHOD_HD_I uint8_t get_color(unsigned int p_side, unsigned int p_orientation) const;
private:
  uint8_t m_colors[4];
};

CUDA_KERNEL(kernel, const piece * const p_pieces, constraint (*p_constraints)[18][4], situation * p_initial_situation, int * p_initial_index)
{
  constraint (&l_constraints)[18][4] = *p_constraints;
  situation l_situation(p_initial_situation[threadIdx.x]);
  constraint l_available_pieces[196];
  int l_max_index = *p_initial_index - 1;
  situation l_max_situation(l_situation);
  uint32_t l_nb_iteration = UINT32_MAX - 10;
  
  // Main loop
  int l_index = *p_initial_index;
  while(l_index < 239 && l_nb_iteration)
    {
      unsigned int l_north_index = l_index - 16;
      unsigned int l_east_index = l_index + 1;
      unsigned int l_south_index = l_index + 16;
      unsigned int l_west_index = l_index - 1;

      // get neighbour piece ids
      uint8_t l_north_piece_id = l_situation.m_piece_ids[l_north_index];
      uint8_t l_east_piece_id = l_situation.m_piece_ids[l_east_index];
      uint8_t l_south_piece_id = l_situation.m_piece_ids[l_south_index];
      uint8_t l_west_piece_id = l_situation.m_piece_ids[l_west_index];

      // get neighbour orientations
      uint32_t l_north_piece_orientation = l_situation.m_orientations.get_orientation(l_north_index);
      uint32_t l_east_piece_orientation = l_situation.m_orientations.get_orientation(l_east_index);
      uint32_t l_south_piece_orientation = l_situation.m_orientations.get_orientation(l_south_index);
      uint32_t l_west_piece_orientation = l_situation.m_orientations.get_orientation(l_west_index);

      // Get colours defining the constraint
      uint8_t l_north_color = p_pieces[l_north_piece_id].get_color((unsigned int)t_orientation::SOUTH,l_north_piece_orientation);
      uint8_t l_east_color = p_pieces[l_east_piece_id].get_color((unsigned int)t_orientation::WEST,l_east_piece_orientation);
      uint8_t l_south_color = p_pieces[l_south_piece_id].get_color((unsigned int)t_orientation::NORTH,l_south_piece_orientation);
      uint8_t l_west_color = p_pieces[l_west_piece_id].get_color((unsigned int)t_orientation::EAST, l_west_piece_orientation);

      // Compute constraint for each orientation
      constraint l_north_constraint(l_constraints[l_north_color][(unsigned int)t_orientation::NORTH],
				    l_constraints[l_east_color][(unsigned int)t_orientation::EAST],
				    l_constraints[l_south_color][(unsigned int)t_orientation::SOUTH],
				    l_constraints[l_west_color][(unsigned int)t_orientation::WEST],
				    l_available_pieces[l_index - 17]
				    );
      constraint l_east_constraint(l_constraints[l_north_color][(unsigned int)t_orientation::EAST],
				   l_constraints[l_east_color][(unsigned int)t_orientation::SOUTH],
				   l_constraints[l_south_color][(unsigned int)t_orientation::WEST],
				   l_constraints[l_west_color][(unsigned int)t_orientation::NORTH],
				   l_available_pieces[l_index - 17]
				   );
      constraint l_south_constraint(l_constraints[l_north_color][(unsigned int)t_orientation::SOUTH],
				    l_constraints[l_east_color][(unsigned int)t_orientation::WEST],
				    l_constraints[l_south_color][(unsigned int)t_orientation::NORTH],
				    l_constraints[l_west_color][(unsigned int)t_orientation::EAST],
				    l_available_pieces[l_index - 17]
				    );
      constraint l_west_constraint(l_constraints[l_north_color][(unsigned int)t_orientation::WEST],
				   l_constraints[l_east_color][(unsigned int)t_orientation::NORTH],
				   l_constraints[l_south_color][(unsigned int)t_orientation::EAST],
				   l_constraints[l_west_color][(unsigned int)t_orientation::SOUTH],
				   l_available_pieces[l_index - 17]
				   );

      
      int l_ffs[4] = {l_north_constraint.ffs(), l_east_constraint.ffs(), l_south_constraint.ffs(), l_west_constraint.ffs()};
      int l_ffs_1_0 = l_ffs[0] ? l_ffs[0] : l_ffs[1];
      unsigned int l_orientation_1_0 = l_ffs[0] ? 0 : 1;
      int l_ffs_1_1 = l_ffs[2] ? l_ffs[2] : l_ffs[3];
      unsigned int l_orientation_1_1 = l_ffs[2] ? 2 : 3;
      int l_ffs_result = l_ffs_1_0 ? l_ffs_1_0 : l_ffs_1_1;
      unsigned int l_orientation_result = l_ffs_1_0 ? l_orientation_1_0 : l_orientation_1_1;

      // Assign piece
      l_situation.m_piece_ids[l_index] = l_ffs_result;
      l_situation.m_orientations.set_orientation(l_index, l_orientation_result);
      l_available_pieces[l_index - 17].toggle_bit(l_situation.m_piece_ids[l_index] - (l_ffs_result ? 1 : 0),l_ffs_result ? true : false);
      int l_backward_increment = ((l_index - 1) & 0xF) ? -1 : -3;
      int l_forward_increment = ((l_index + 2) & 0xF) ? 1 : 3;
      l_available_pieces[l_index - 17 + l_forward_increment] = l_available_pieces[l_index - 17];
      l_index += l_ffs_result ? l_forward_increment : l_backward_increment;
      if(l_index > l_max_index)
	{
	  l_max_situation = l_situation;
	  l_max_index = l_index;
	}
      ++l_nb_iteration;
    }
  p_initial_situation[threadIdx.x] = l_max_situation;

}

//------------------------------------------------------------------------------
int main(void)
{
  test_constraint();
  test_orientation();

  // Binary representation of pieces
  piece l_pieces[197];

  uint32_t l_center_id_to_piece_id[196];

  // Binary representation of constraints by colors
  // Color 0 represent no pieces
  constraint l_constraints[18][4];
  for(unsigned int l_index = 0; l_index < 4 ; ++l_index)
    {
      l_constraints[0][l_index].fill(true);
    }

  unsigned int l_color_id_to_center_color_id[23] = 
    {
      256,
      256,
      256,
      256,
      256,
      256,
      256,
      256,
      256,
      256,
      256,
      256,
      256,
      256,
      256,
      256,
      256,
      256,
      256,
      256,
      256,
      256,
      256
    };

  unsigned int l_center_color_id_to_color_id[18] = 
    {
      128,
      128,
      128,
      128,
      128,
      128,
      128,
      128,
      128,
      128,
      128,
      128,
      128,
      128,
      128,
      128,
      128,
      128
    };

  // Compute pieces bitfield representation
  unsigned int l_center_piece_index = 1;

  unsigned int l_center_color_index = 1;

#include "eternity2_pieces.h"

  for(unsigned int l_all_pieces_index = 0; l_all_pieces_index < 256; ++l_all_pieces_index)
    {
      if(0 != l_all_pieces[l_all_pieces_index][1 + (unsigned int)t_orientation::NORTH] &&
	 0 != l_all_pieces[l_all_pieces_index][1 + (unsigned int)t_orientation::EAST] &&
	 0 != l_all_pieces[l_all_pieces_index][1 + (unsigned int)t_orientation::SOUTH] &&
	 0 != l_all_pieces[l_all_pieces_index][1 + (unsigned int)t_orientation::WEST]
	 )
	{
	  for(unsigned int l_orientation_index = (unsigned int)t_orientation::NORTH; l_orientation_index <= (unsigned int)t_orientation::WEST; ++l_orientation_index)
	    {
	      unsigned int l_color_id = l_all_pieces[l_all_pieces_index][1 + l_orientation_index];

	      // Compute center_color_id if not already done
	      if(256 == l_color_id_to_center_color_id[l_color_id])
		{
		  l_color_id_to_center_color_id[l_color_id] = l_center_color_index;
		  l_center_color_id_to_color_id[l_center_color_index] = l_color_id;
		  ++l_center_color_index;
		}
	      unsigned int l_center_color_id = l_color_id_to_center_color_id[l_color_id];
	      // Store bitfield representation
	      l_pieces[l_center_piece_index].set_color(l_center_color_id, l_orientation_index);

	      // Record this color in constraint table. We consider l_center_piece_index - 1 because piece 0 mean no piece
	      l_constraints[l_center_color_id][l_orientation_index].set_bit(l_center_piece_index - 1);
	    }

	  // Keep memory of global piece id
	  l_center_id_to_piece_id[l_center_piece_index] = l_all_pieces[l_all_pieces_index][0];

	  ++l_center_piece_index;
	}
    }

  int l_initial_index = 17;

  situation l_initial_situation;

  l_initial_situation.m_piece_ids[1] = 1;
  l_initial_situation.m_piece_ids[2] = 77;
  l_initial_situation.m_piece_ids[18] = 1;
  l_initial_situation.m_piece_ids[16] = 1;
  l_initial_situation.m_piece_ids[33] = 1;
  l_initial_situation.m_orientations.set_orientation(1,(unsigned int)t_orientation::SOUTH);
  l_initial_situation.m_orientations.set_orientation(18,(unsigned int)t_orientation::SOUTH);
  l_initial_situation.m_orientations.set_orientation(16,(unsigned int)t_orientation::SOUTH);
  l_initial_situation.m_orientations.set_orientation(33,(unsigned int)t_orientation::SOUTH);

  piece * l_pieces_ptr = nullptr;
  constraint (*l_constraints_ptr)[18][4] = nullptr;
  situation * l_initial_situation_ptr = nullptr;
  int * l_initial_index_ptr = nullptr;

  gpuErrChk(hipMalloc(&l_pieces_ptr, 197 * sizeof(piece)));
  gpuErrChk(hipMalloc(&l_constraints_ptr, 18 * 4 * sizeof(constraint)));
  gpuErrChk(hipMalloc(&l_initial_situation_ptr, sizeof(situation)));
  gpuErrChk(hipMalloc(&l_initial_index_ptr, sizeof(int)));
  
  gpuErrChk(hipMemcpy(l_pieces_ptr, &l_pieces[0], 197 * sizeof(piece), hipMemcpyHostToDevice));
  gpuErrChk(hipMemcpy(l_constraints_ptr, &l_constraints[0][0], 18 * 4 * sizeof(constraint), hipMemcpyHostToDevice));
  gpuErrChk(hipMemcpy(l_initial_situation_ptr, &l_initial_situation, sizeof(situation), hipMemcpyHostToDevice));
  gpuErrChk(hipMemcpy(l_initial_index_ptr, &l_initial_index, sizeof(int), hipMemcpyHostToDevice));
  
  dim3 dimBlock(1,1);
  dim3 dimGrid(1,1);
  launch_kernels(kernel, dimGrid, dimBlock, l_pieces_ptr,
		 l_constraints_ptr,
		 l_initial_situation_ptr,
		 l_initial_index_ptr
		 );

  gpuErrChk(hipMemcpy(&l_initial_situation, l_initial_situation_ptr, sizeof(situation), hipMemcpyDeviceToHost));
  gpuErrChk(hipFree(l_pieces_ptr));
  gpuErrChk(hipFree(l_constraints_ptr));
  gpuErrChk(hipFree(l_initial_situation_ptr));
  gpuErrChk(hipFree(l_initial_index_ptr));
//#endif

  return 0;
}

//------------------------------------------------------------------------------
constraint::constraint(bool p_bool):
  m_words
   {
     p_bool ? 0xFFFFFFFF : 0,
     p_bool ? 0xFFFFFFFF : 0,
     p_bool ? 0xFFFFFFFF : 0,
     p_bool ? 0xFFFFFFFF : 0,
     p_bool ? 0xFFFFFFFF : 0,
     p_bool ? 0xFFFFFFFF : 0,
     p_bool ? 0xFFFFFFFF : 0
   }
{
}

//------------------------------------------------------------------------------
void constraint::fill(bool p_bool)
{
  memset(m_words, p_bool ? 0xFF : 0x0, 7 * sizeof(uint32_t));
}

//------------------------------------------------------------------------------
constraint::constraint(const constraint & p_c1,
		       const constraint & p_c2,
		       const constraint & p_c3,
		       const constraint & p_c4,
		       const constraint & p_c5
		       ):
  m_words{p_c1.m_words[0] & p_c2.m_words[0] & p_c3.m_words[0] & p_c4.m_words[0] & p_c5.m_words[0],
          p_c1.m_words[1] & p_c2.m_words[1] & p_c3.m_words[1] & p_c4.m_words[1] & p_c5.m_words[1],
          p_c1.m_words[2] & p_c2.m_words[2] & p_c3.m_words[2] & p_c4.m_words[2] & p_c5.m_words[2],
          p_c1.m_words[3] & p_c2.m_words[3] & p_c3.m_words[3] & p_c4.m_words[3] & p_c5.m_words[3],
          p_c1.m_words[4] & p_c2.m_words[4] & p_c3.m_words[4] & p_c4.m_words[4] & p_c5.m_words[4],
          p_c1.m_words[5] & p_c2.m_words[5] & p_c3.m_words[5] & p_c4.m_words[5] & p_c5.m_words[5],
          p_c1.m_words[6] & p_c2.m_words[6] & p_c3.m_words[6] & p_c4.m_words[6] & p_c5.m_words[6]
         }
{
}

//------------------------------------------------------------------------------
void constraint::set_bit(uint32_t p_index)
{
  m_words[ p_index >> 5] |= (1 << (p_index & 0x1F));
}

//------------------------------------------------------------------------------
void constraint::unset_bit(uint32_t p_index)
{
  m_words[ p_index >> 5] &= ~(1 << (p_index & 0x1F));
}

//------------------------------------------------------------------------------
void constraint::toggle_bit(uint32_t p_index, bool p_value)
{
  m_words[ p_index >> 5] ^= (((uint32_t)p_value) << (p_index & 0x1F));
}

//------------------------------------------------------------------------------
bool constraint::get_bit(uint32_t p_index) const
{
  return m_words[ p_index >> 5] & (1 << (p_index & 0x1F));
}

//------------------------------------------------------------------------------
int constraint::ffs(void) const
{
  int l_ffs[7];
  for(int l_index = 0; l_index < 7; ++l_index)
    {
#ifdef __CUDA_ARCH__
      l_ffs[l_index] = __ffs(m_words[l_index]) + 32 * l_index;
#else
      l_ffs[l_index] = ::ffs(m_words[l_index]) + 32 * l_index;
#endif
    }
  int l_ffs_1_0 = l_ffs[0] ? l_ffs[0] : l_ffs[1];
  int l_ffs_1_1 = l_ffs[2] ? l_ffs[2] : l_ffs[3];
  int l_ffs_1_2 = l_ffs[4] ? l_ffs[4] : l_ffs[5];

  int l_ffs_2_0 = l_ffs_1_0 ? l_ffs_1_0 : l_ffs_1_1;
  int l_ffs_2_1 = l_ffs_1_2 ? l_ffs_1_2 : l_ffs[6];
  return l_ffs_2_0 ? l_ffs_2_0 : l_ffs_2_1;
}

//------------------------------------------------------------------------------
void test_constraint(void)
{
  std::cout << "Start test_constraint" << std::endl ;
  constraint l_constraint;
  for(unsigned int l_index = 0 ; l_index < 196 ; ++l_index)
    {
      if(l_constraint.get_bit(l_index))
	{
	  std::stringstream l_stream;
	  l_stream << l_index;
	  throw std::logic_error("Bit[" + l_stream.str() + "] should be zero");
	}
    }

  for(unsigned int l_tested_index = 0; l_tested_index < 196; ++l_tested_index)
    {
      l_constraint.set_bit(l_tested_index);
      for(unsigned int l_index = 0; l_index < 196; ++l_index)
	{
	  bool l_expected_result = l_index == l_tested_index;
	  if(l_expected_result != l_constraint.get_bit(l_index))
	    {
	      std::stringstream l_stream;
	      l_stream << l_index;
	      throw std::logic_error("Bit[" + l_stream.str() + "] should be " + (l_expected_result ? "true" : "false"));
	    }
	}
      l_constraint.unset_bit(l_tested_index);
    }
  std::cout << "test_constraint OK" << std::endl ;
}

//------------------------------------------------------------------------------
situation_orientation::situation_orientation(void)
{
  memset(&m_orientations[0], 0, 64);
}

//------------------------------------------------------------------------------
situation_orientation::situation_orientation(const situation_orientation & p_orientation)
{
  memcpy(&m_orientations[0], & p_orientation.m_orientations[0], 16 * sizeof(uint32_t));
}

//------------------------------------------------------------------------------
void situation_orientation::set_orientation(unsigned int p_index,
					    uint32_t p_orientation)
{
  assert(p_orientation <= 3);
  unsigned int l_shift = (p_index & 0xF) << 1;
  unsigned int l_index = p_index >> 4;
  m_orientations[l_index] &= ~(((uint32_t)0x3) << l_shift);
  m_orientations[l_index] |= (p_orientation << l_shift);
}

//------------------------------------------------------------------------------
void situation_orientation::set_orientation(unsigned int p_x_index,
					    unsigned int p_y_index,
					    uint32_t p_orientation)
{
  unsigned int l_index = (p_y_index << 4) | p_x_index;
  set_orientation(l_index, p_orientation);
}

//------------------------------------------------------------------------------
uint32_t situation_orientation::get_orientation(unsigned int p_index) const
{
  unsigned int l_shift = (p_index & 0xF) << 1;
  unsigned int l_index = p_index >> 4;
  return (m_orientations[l_index] >> l_shift) & 0x3; 
}

//------------------------------------------------------------------------------
uint32_t situation_orientation::get_orientation(unsigned int p_x_index,
						unsigned int p_y_index) const
{
  unsigned int l_index = (p_y_index << 4) | p_x_index;
  return get_orientation(l_index);
}

//------------------------------------------------------------------------------
void test_orientation(void)
{
  std::cout << "Start test_orientation" << std::endl ;
  situation_orientation l_orientation;
  for(unsigned int l_index = 0 ; l_index < 256 ; ++l_index)
    {
      if(l_orientation.get_orientation(l_index))
	{
	  std::stringstream l_stream;
	  l_stream << l_index;
	  throw std::logic_error("orientation[" + l_stream.str() + "] should be zero");
	}
    }

  for(unsigned int l_orientation_value = 0; l_orientation_value < 4; ++l_orientation_value)
    {
      std::stringstream l_value_stream;
      l_value_stream << l_orientation_value;
      for(unsigned int l_tested_index = 0; l_tested_index < 256; ++l_tested_index)
	{
	  std::stringstream l_tested_stream;
	  l_tested_stream << l_tested_index;
	  l_orientation.set_orientation(l_tested_index, l_orientation_value);
	  for(unsigned int l_index = 0; l_index < 256; ++l_index)
	    {
	      unsigned int l_expected_result = l_index == l_tested_index ? l_orientation_value : 0;
	      unsigned int l_result = l_orientation.get_orientation(l_index);
	      if(l_expected_result != l_result)
		{
		  std::stringstream l_stream;
		  l_stream << l_index;
		  std::stringstream l_expected_stream;
		  l_expected_stream << l_expected_result;
		  std::stringstream l_result_stream;
		  l_result_stream << l_result;
		  throw std::logic_error("tested[" + l_tested_stream.str() + "] = " + l_value_stream.str() + ": orientation[" + l_stream.str() + "] value (" + l_result_stream.str() + ") is not the expected value (" + l_expected_stream.str() + ")");
		}
	    }
	  l_orientation.set_orientation(l_tested_index, 0);
	}
    }
  std::cout << "test_orientation OK" << std::endl ;
}

//------------------------------------------------------------------------------
situation::situation(const situation & p_situation)
{
  memcpy(&m_orientations, &p_situation.m_orientations, sizeof(m_orientations));
  memcpy(&m_piece_ids, &p_situation.m_piece_ids, sizeof(m_piece_ids));
}

//------------------------------------------------------------------------------
void situation::operator=(const situation & p_situation)
{
  memcpy(&m_orientations, &p_situation.m_orientations, sizeof(m_orientations));
  memcpy(&m_piece_ids, &p_situation.m_piece_ids, sizeof(m_piece_ids));
}

//------------------------------------------------------------------------------
piece::piece(void):
  m_colors{0,0,0,0}
{
}

//------------------------------------------------------------------------------
void piece::set_color(uint8_t p_color,
		      unsigned int p_orientation)
{
  m_colors[p_orientation] = p_color;
}

//------------------------------------------------------------------------------
uint8_t piece::get_color(unsigned int p_side) const
{
  assert(p_side <= 3);
  return m_colors[p_side];
}

//------------------------------------------------------------------------------
uint8_t piece::get_color(unsigned int p_side, unsigned int p_orientation) const
{
  return get_color((p_side + p_orientation) & 0x3);
}
// EOF
